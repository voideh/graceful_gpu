
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <iostream>
#include <algorithm>
using namespace std;

int factorial(int n)
{
  return (n == 0 || n == 1) ? 1 : factorial(n-1) * n;
}

__global__ void calculate_edges(int *perms, int *children, int *stops, int *edges, int NUMNODES, int NUMPERMS)
{
    /*
       Since the permutation array is a flattened 2D array which was NUMNODES wide and NUMPERMS long, then we
       must start at the begining of every row which would be offset by NUMNODES.
   */
    int element = (blockIdx.x * blockDim.x + threadIdx.x) * NUMNODES; 
    int total = NUMNODES * NUMPERMS; // make sure we do not exceed the size of the permutation array
    int edge_counter = 0; // keep track of where in the edge array we are putting the next edge label
    int last_index = 0; // keep track of the last index from the stop array.
    int edge_start = (blockIdx.x * blockDim.x + threadIdx.x) * (NUMNODES-1); // calculate where in the edge array we should begin placing labels
    if(element < total)
    {
        // Only go thorugh each NUMNODE group of labels
        for(int i = element; i < element + NUMNODES; i++)
        {
            // check for sentinel value of -1
            if(stops[i % NUMNODES] != -1)
            {
                // If this is our first time we start at 0, otherwise we continue from the the last index
                for(int j = (last_index == 0) ? 0 : last_index+1; j<=stops[i % NUMNODES]; j++)
                    // place the absolute difference of each end point into the edge array
                    edges[edge_start + edge_counter++] = abs(perms[i] - perms[children[j] + element]);
                last_index = stops[i % NUMNODES];
            }
        } }
}

__global__ void check_gracefulness(int *edges, int *graceful_labels, int NUMNODES, int NUMPERMS)
{
    /*
       Go through edge array and check for any duplicates. If there are duplicates found, exit the loop and mark this label
       as being nongraceful , which is designated by a -1 in the label array. If no duplicates are found, the labeling is graceful and
       the index of the permutation is stored.
   */
    int element = (blockIdx.x * blockDim.x + threadIdx.x) * (NUMNODES-1); 
    int total = NUMNODES * NUMPERMS;
    bool graceful = true;
    if(element < total)
    {
        for(int i = element; i < element + NUMNODES-1; i++)
        {
            int current = edges[i];
            for(int j = i + 1; j < element + NUMNODES-1; j++)
            {
                if(current == edges[j])
                {
                    graceful = false;
                    break;
                }
            }
            if(!graceful) break;
        }
        if(graceful)
            graceful_labels[element / (NUMNODES-1)] = element;
        if(!graceful)
            graceful_labels[element / (NUMNODES-1)] = -1;
    }
}

void execute_gpu(int perms[], int children[], int stops[], int graceful_labels[], int edges[], int NUMNODES, int NUMPERMS)
{
    int *d_perms, *d_children, *d_graceful_labels, *d_stops, *d_edges;

    // define sizes for convenience
    const size_t perm_size = NUMNODES*NUMPERMS*sizeof(int);
    const size_t edge_size = (NUMNODES-1)*NUMPERMS*sizeof(int);
    const size_t child_size = (NUMNODES-1)*sizeof(int);
    const size_t stop_size = NUMNODES*sizeof(int);
    const size_t label_size = NUMPERMS*sizeof(int);

    // 768 cores available on my home computer
    // 1024 cores available on starship
    int numCores = (NUMNODES * NUMPERMS)/ 1024 + 1;
    int numThreads = 1024;

    // Allocate memory on GPU
    hipMalloc(&d_perms, perm_size);
    hipMalloc(&d_edges, edge_size);
    hipMalloc(&d_children, child_size);
    hipMalloc(&d_stops, stop_size);
    hipMalloc(&d_graceful_labels, label_size);

    // Copy over necessary arrays to GPU
    hipMemcpy(d_perms, perms, perm_size, hipMemcpyHostToDevice);
    hipMemcpy(d_children, children, child_size, hipMemcpyHostToDevice);
    hipMemcpy(d_stops, stops, stop_size, hipMemcpyHostToDevice);

    // Calculate edge labelings for each permutation
    calculate_edges<<<numCores, numThreads>>>(d_perms,
            d_children,
            d_stops,
            d_edges,
            NUMNODES,
            NUMPERMS);

    // Don't need these for the next step, so just free the memory up.
    hipFree(&d_perms);
    hipFree(&d_stops);
    hipFree(&d_children);

    // For debugging  purposes only
     hipMemcpy(edges, d_edges, edge_size, hipMemcpyDeviceToHost); 

    // Now check the gracefulness of the given edge labelings.
    check_gracefulness<<<numCores, numThreads>>>(d_edges, d_graceful_labels, NUMNODES, NUMPERMS);

    // Copy back the evaluated labelings
    hipMemcpy(graceful_labels, d_graceful_labels, label_size, hipMemcpyDeviceToHost);

    // Free up the rest of the memory
    hipFree(&d_graceful_labels);
    hipFree(&d_edges);
}

int main()
{
    //const int NUMNODES = 5;
    //const int MAX_PERMS = factorial(5);
    //int stops [] = {1,-1,3,-1,-1};
    const int NUMNODES = 3;
    const int MAX_PERMS = factorial(NUMNODES);
    int stops [] = {1,-1,-1};
    int children[NUMNODES-1],  labels[NUMNODES];
    int found = 0;
// 11 node tree
   //int stops [] = {1,2,5,6,7,8,9,-1,-1,-1,-1};
    bool has_next = false;
    // generate both children and label array
    for(int i = 0; i < NUMNODES; i++)
    {
        labels[i] = i;
        if(i < (NUMNODES - 1)) children[i] = i+1;
    }
//do{
	int perms[MAX_PERMS*NUMNODES], graceful_labels[MAX_PERMS], edges[MAX_PERMS*(NUMNODES-1)];
        for(int i = 0; i < MAX_PERMS; i++)
        {
            for(int j = 0; j < NUMNODES; j++)
            {
                perms[i*NUMNODES+j] = labels[j];
            }
            graceful_labels[i] = -1;
		 next_permutation(labels, labels+NUMNODES);
        }
        execute_gpu(perms, children, stops, graceful_labels, edges, NUMNODES, MAX_PERMS);

        for(int i = 0; i < MAX_PERMS; i++)
        {
            if(graceful_labels[i] != -1)
		{
                found++;
		}
	}

for(int i = 0; i < MAX_PERMS; i++)
{
for(int j = 0; j < NUMNODES; j++)
{
	cout << perms[i*NUMNODES+j] << " " ;
}
cout << endl;
for(int j = 0; j < NUMNODES-1; j++)
	cout << edges[i*NUMNODES+j] << " ";
cout << endl;
}

//}while(has_next);
    cout << "Found " << found << " graceful labelings." << endl;
    return 0;
}
