
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <iostream>
#include <algorithm>
using namespace std;

int factorial(int n)
{
  return (n == 0 || n == 1) ? 1 : factorial(n-1) * n;
}

__global__ void calculate_edges(int *perms, int *children, int *stops, int *graceful, int *temp, int NUMNODES, int NUMPERMS)
{
    // will start at begining of node permutation
    int element = (blockIdx.x * blockDim.x + threadIdx.x) * NUMNODES; 
    int total = NUMNODES * NUMPERMS;
    int temp_counter = 0;
    int last_index = 0;
    if(element < total)
    {
        for(int i = element; i < element + NUMNODES; i++)
        {
            if(stops[i % NUMNODES] != -1)
            {
                for(int j = (last_index == 0) ? 0 : last_index+1; j<=stops[i % NUMNODES]; j++)
                    temp[element + temp_counter++] = abs(perms[i] - perms[children[j] + element]);
                last_index = stops[i % NUMNODES];
            }
        }
    }
}

void execute_gpu(int perms[], int children[], int stops[], int graceful_labels[], int temp[], int NUMNODES, int NUMPERMS)
{
    int *d_perms, *d_children, *d_graceful_labels, *d_stops, *temp_calc;

    const size_t perm_size = NUMNODES*NUMPERMS*sizeof(int);
    const size_t child_size = (NUMNODES-1)*sizeof(int);
    const size_t stop_size = NUMNODES*sizeof(int);
    const size_t label_size = NUMPERMS*sizeof(int);

    int numCores = (NUMNODES * NUMPERMS)/ 768 + 1;
    int numThreads = 1024;

    hipMalloc(&d_perms, perm_size);
    hipMalloc(&temp_calc, perm_size);
    hipMalloc(&d_children, child_size);
    hipMalloc(&d_stops, stop_size);
    hipMalloc(&d_graceful_labels, label_size);

    hipMemcpy(d_perms, perms, perm_size, hipMemcpyHostToDevice);
    hipMemcpy(d_children, children, child_size, hipMemcpyHostToDevice);
    hipMemcpy(d_stops, stops, stop_size, hipMemcpyHostToDevice);

    calculate_edges<<<numCores, numThreads>>>(d_perms,
            d_children,
            d_stops,
            d_graceful_labels,
            temp_calc,
            NUMNODES,
            NUMPERMS);
    hipMemcpy(temp, temp_calc, perm_size, hipMemcpyDeviceToHost);

    hipFree(&d_perms);
    hipFree(&d_stops);
    hipFree(&d_children);
    hipFree(&d_graceful_labels);
    hipFree(&temp_calc);
}

int main()
{
    const int NUMNODES = 5;
    const int NUMPERMS = factorial(NUMNODES);
    int children[NUMNODES-1], perms[NUMPERMS*NUMNODES], graceful_labels[NUMPERMS], labels[NUMNODES];
    int temp[NUMPERMS*NUMNODES-1];
    int stops[] = {1, 3, -1, -1, -1};
    // generate both children and label array
    for(int i = 0; i < NUMNODES; i++)
    {
        labels[i] = i;
        if(i < NUMNODES - 1) children[i] = i+1;
    }

    // create all permutations of given nodes
    for(int i = 0; i < NUMPERMS; i++)
    {
        for(int j = 0; j < NUMNODES; j++)
        {
            perms[i*NUMNODES+j] = labels[j];
            temp[i*NUMNODES+j] = 0;
        }
        graceful_labels[i] = -1;
        next_permutation(labels, labels+NUMNODES);
    }
    execute_gpu(perms, children, stops, graceful_labels, temp, NUMNODES, NUMPERMS);

    for(int i = 0; i < NUMPERMS; i++)
    {
        for(int j = 0; j < NUMNODES-1; j++)
            cout << temp[i*NUMNODES+j] << " ";
        cout << endl;
    }
}
