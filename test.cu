#include "hip/hip_runtime.h"
#include <iostream>


__global__ void sqrt(int n)
{
    int test[n];
    for(int i = 0; i < n; i++)
        test[i] = i * i;
}

void gpu(int N)
{
    int numThreads = 1024;
    int numCores = N/768 + 1;
    int* gpu;
   hipMalloc(&gpu, N*sizeof(float)); // Allocate enough memory on the GPU
   sqrt<<numCores, numThreads>>(N);
   hipFree(&gpu);
}

int main()
{
    gpu(50);
    return 0;
}
